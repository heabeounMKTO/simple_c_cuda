#include "hip/hip_runtime.h"
#include "vec.h"
#include "vec_gpu.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <stdlib.h>

__device__ void vec_add_device(float* v1, float* v2, float* result, int idx) {
    result[idx] = v1[idx] + v2[idx];
}

__global__ void vec_add_kernel(float* v1, float* v2, float* result, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        vec_add_device(v1, v2, result, idx);
    }
}

void vec_add_cu(const vec_N* v1, const vec_N* v2, vec_N* result) {
  size_t components_memsize = v1->dimension * sizeof(float);
  float *d_v1 , *d_v2, *d_res;   

  if (hipMalloc(&d_v1, components_memsize) != hipSuccess ||
      hipMalloc(&d_v2, components_memsize)  != hipSuccess ||
      hipMalloc(&d_res, components_memsize) != hipSuccess) {
    fprintf(stderr, "CUDA memory allocation failed\n");
  }

  if (hipMemcpy(d_v1, v1->components, components_memsize, hipMemcpyHostToDevice) != hipSuccess ||
      hipMemcpy(d_v2, v2->components, components_memsize, hipMemcpyHostToDevice) != hipSuccess)
  {
    fprintf(stderr, "CUDA memory copy to device failed\n");
  }

  int threadsPerBlock = 256;
  int blocksPerGrid = (v1->dimension + threadsPerBlock - 1) / threadsPerBlock;
  vec_add_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_v1, d_v2, d_res, v1->dimension);
  
  hipError_t kernelError = hipGetLastError();

  if (kernelError != hipSuccess) {
    fprintf(stderr, "kernel launch error: %s\n",
            hipGetErrorString(kernelError));
  }

  if (hipMemcpy(result->components, d_res, components_memsize, hipMemcpyDeviceToHost) !=
      hipSuccess) 
  {
    hipError_t cpy_error = hipGetLastError();
    fprintf(stderr, "CUDA memory copy to host failed\n");
    fprintf(stderr, "device to host Copy error: %s\n",
            hipGetErrorString(cpy_error));
  }
  
  hipFree(d_v1);
  hipFree(d_v2);
  hipFree(d_res);

  
}
